#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>

#define CUDA_CHECK(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      std::fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n",   \
                   err, hipGetErrorString(err));                              \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

__global__ void matrixVectorMultiplicationKernel(float *output, const float *X, const float *y, const int width) {
  auto outputIndex = blockIdx.x * blockDim.x + threadIdx.x;
  if (outputIndex < width) {
    float outputElement = 0.0;
    for (auto index = 0 ; index < width; ++index) {
        auto xIndex = outputIndex * width + index;
        auto yIndex = index;
        outputElement += (X[xIndex] * y[yIndex]);
    }
    output[outputIndex] = outputElement;
  }
}

void matrixVectorMultiplication(float *output, float *X, float *y, const int width) {
  int vectorSize = width * sizeof(float);
  int matrixSize = width * width * sizeof(float);
  float *d_output, *d_X, *d_y;

  CUDA_CHECK(hipMalloc(&d_output, vectorSize));
  CUDA_CHECK(hipMalloc(&d_X, matrixSize));
  CUDA_CHECK(hipMalloc(&d_y, vectorSize));

  CUDA_CHECK(hipMemcpy(d_X, X, matrixSize, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_y, y, vectorSize, hipMemcpyHostToDevice));

  dim3 gridDim(std::ceil(width / 16.0));
  dim3 blockDim(16);
  matrixVectorMultiplicationKernel<<<gridDim, blockDim>>>(d_output, d_X, d_y, width);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(output, d_output, vectorSize, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_output));
  CUDA_CHECK(hipFree(d_X));
  CUDA_CHECK(hipFree(d_y));
}
