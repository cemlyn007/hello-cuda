#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>

#define CUDA_CHECK(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      std::fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n",   \
                   err, hipGetErrorString(err));                              \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

__global__ void matrixMultiplicationKernel(float *Output, const float *X, const float *Y, const int width) {
  auto row = blockIdx.x * blockDim.x + threadIdx.x;
  auto column = blockIdx.y * blockDim.y + threadIdx.y;
  auto outputIndex = row * width + column;
  if (row < width && column < width) {
    for (auto index = 0; index < width; ++index) {
      auto xIndex = row * width + index;
      auto yIndex = index * width + column;
      Output[outputIndex] += (X[xIndex] * Y[yIndex]);
    }
  }
}

void matrixMultiplication(float *Output, float *X, float *Y, const int width) {
  int size = width * width * sizeof(float);
  float *d_Output, *d_X, *d_Y;

  CUDA_CHECK(hipMalloc(&d_Output, size));
  CUDA_CHECK(hipMalloc(&d_X, size));
  CUDA_CHECK(hipMalloc(&d_Y, size));

  CUDA_CHECK(hipMemcpy(d_X, X, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice));

  dim3 gridDim(std::ceil(width / 16.0), std::ceil(width / 16.0));
  dim3 blockDim(16, 16);
  matrixMultiplicationKernel<<<gridDim, blockDim>>>(d_Output, d_X, d_Y, width);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(Output, d_Output, size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_Output));
  CUDA_CHECK(hipFree(d_X));
  CUDA_CHECK(hipFree(d_Y));
}
