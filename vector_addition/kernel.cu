#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>

#define CUDA_CHECK(expr)                                                     \
  do {                                                                       \
    hipError_t err = (expr);                                                \
    if (err != hipSuccess) {                                                \
      std::fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n", \
              err, hipGetErrorString(err));                                 \
      exit(err);                                                             \
    }                                                                        \
  } while (0)

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vecAdd(float* A, float* B, float* C, int n) {
  int size = n * sizeof(float);
  float *d_A, *d_B, *d_C;

  CUDA_CHECK(hipMalloc(&d_A, size));
  CUDA_CHECK(hipMalloc(&d_B, size));
  CUDA_CHECK(hipMalloc(&d_C, size));

  CUDA_CHECK(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

  vecAddKernel<<<std::ceil(n / 256.0), 256.0>>>(d_A, d_B, d_C, n);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_A));
  CUDA_CHECK(hipFree(d_B));
  CUDA_CHECK(hipFree(d_C));
}
