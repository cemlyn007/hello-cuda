#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>

#define CUDA_CHECK(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      std::fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n",   \
                   err, hipGetErrorString(err));                              \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

// Note that I am assuming RGB, and have not implemented support for RGBA.
__global__ static const int CHANNELS = 3;

__global__ void rgbToGrayscaleKernel(unsigned char *Pout, unsigned char *Pin,
                                     int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (col < width && row < height) {
    int grayOffset = row * width + col;

    unsigned char red = Pin[grayOffset * CHANNELS];
    unsigned char green = Pin[grayOffset * CHANNELS + 1];
    unsigned char blue = Pin[grayOffset * CHANNELS + 2];

    Pout[grayOffset] = 0.21 * red + 0.72 * green + 0.07 * blue;
  }
}

void rgbToGrayscale(unsigned char *Pout, unsigned char *Pin, int width,
                    int height) {
  int sizeOut = width * height * sizeof(unsigned char);
  int sizeIn = sizeOut * CHANNELS;
  unsigned char *d_Pout, *d_Pin;

  CUDA_CHECK(hipMalloc(&d_Pout, sizeOut));
  CUDA_CHECK(hipMalloc(&d_Pin, sizeIn));

  CUDA_CHECK(hipMemcpy(d_Pin, Pin, sizeIn, hipMemcpyHostToDevice));

  dim3 gridDim(std::ceil(width / 16.0), std::ceil(height / 16.0), 1);
  dim3 blockDim(16, 16, 1);
  rgbToGrayscaleKernel<<<gridDim, blockDim>>>(d_Pout, d_Pin, width, height);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(Pout, d_Pout, sizeOut, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_Pout));
  CUDA_CHECK(hipFree(d_Pin));
}
