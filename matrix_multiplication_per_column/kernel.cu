#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>

#define CUDA_CHECK(expr)                                                       \
  do {                                                                         \
    hipError_t err = (expr);                                                  \
    if (err != hipSuccess) {                                                  \
      std::fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n",   \
                   err, hipGetErrorString(err));                              \
      exit(err);                                                               \
    }                                                                          \
  } while (0)

__global__ void matrixMultiplicationKernel(float *Output, const float *X, const float *Y, const int width) {
  auto column = blockIdx.x * blockDim.x + threadIdx.x;
  if (column < width) {
    for (auto row = 0; row < width; ++row) {
      auto outputIndex = column * width + row;
      for (auto index = 0; index < width; ++index) {
        auto xIndex = column * width + index;
        auto yIndex = index * width + row;
        Output[outputIndex] += (X[xIndex] * Y[yIndex]);
      }
    }
  }
}

void matrixMultiplication(float *Output, float *X, float *Y, const int width) {
  int size = width * width * sizeof(float);
  float *d_Output, *d_X, *d_Y;

  CUDA_CHECK(hipMalloc(&d_Output, size));
  CUDA_CHECK(hipMalloc(&d_X, size));
  CUDA_CHECK(hipMalloc(&d_Y, size));

  CUDA_CHECK(hipMemcpy(d_X, X, size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_Y, Y, size, hipMemcpyHostToDevice));

  dim3 gridDim(std::ceil(width / 16.0));
  dim3 blockDim(16);
  matrixMultiplicationKernel<<<gridDim, blockDim>>>(d_Output, d_X, d_Y, width);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipMemcpy(Output, d_Output, size, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_Output));
  CUDA_CHECK(hipFree(d_X));
  CUDA_CHECK(hipFree(d_Y));
}
